
#include <hip/hip_runtime.h>
#include <tuple>

#ifndef SAFE_FREE
#define SAFE_FREE(MPTR) [MPTR] { if (MPTR != nullptr) { free(MPTR); } }()
#endif

#ifndef SAFE_CUDA_FREE
#define SAFE_CUDA_FREE(MPTR) [MPTR] { if (MPTR != nullptr) { hipFree(MPTR); } }()
#endif

inline int32_t CEIL_DIV(const int32_t num, const int32_t div) {
    return (num + div - 1) / div;
}


template <typename T>
T* deviceTensorRand(int batches, int rows, int columns, float randScale = 1.0f, T** optOutCpuPtr = nullptr) {
    int32_t resultElements = batches * rows * columns;
    int32_t resultSizeInBytes = resultElements * sizeof(T);

    T* device = nullptr;
    T* host = reinterpret_cast<T*>(malloc(resultSizeInBytes));
    if (host != nullptr) {
        if (hipMalloc(&device, resultSizeInBytes) == hipSuccess) {
            for (int32_t i=0; i<resultElements; ++i) {
                float floatValue = (2.0 * std::rand() / (float)RAND_MAX) - 1.0;
                T value = static_cast<T>(floatValue * randScale);
                host[i] = value;
            }
            hipMemcpy(device, host, resultSizeInBytes, hipMemcpyHostToDevice);
        }
        if (optOutCpuPtr != nullptr) {
            *optOutCpuPtr = host;
        } else {
            SAFE_FREE(host);
        }
    }

    return device;
}

template <typename T>
void printTensor(const char* labelName, T* tensor, int32_t rows, int32_t columns) {
    printf("%s", labelName);
    printTensor(tensor, rows, columns);
}

template <typename T>
void printTensor(T* tensor, int32_t rows, int32_t columns) {
    for (int32_t i=0; i<rows; ++i) {
        printf("[");
        for (int32_t j=0; j < columns; ++j) {
            printf("% 7.2f", static_cast<float>(tensor[i * columns + j]));
        }
        printf(" ]\n");
    }
    printf("\n");
}

template <typename T>
std::tuple<uint32_t, double> debugCompare(T* cpuTensorPtr, T* gpuTensorPtr, T** optGpuTensorCpuPtr,
    int32_t numElements, float EPSILON = 0.001f, bool printDeltas = false) {
    int32_t sizeInBytes = numElements * sizeof(T);
    T* gpuTensorCpuMapped = reinterpret_cast<T*>(malloc(sizeInBytes));
    hipMemcpy(gpuTensorCpuMapped, gpuTensorPtr, sizeInBytes, hipMemcpyDeviceToHost);

    double mse = 0.0;
    uint32_t diffCount = 0;
    for (int i=0; i<numElements; i++) {
        float cpuVal = static_cast<float>(cpuTensorPtr[i]);
        float gpuVal = static_cast<float>(gpuTensorCpuMapped[i]);
        mse += pow(cpuVal - gpuVal, 2);
        if (fabs(cpuVal - gpuVal) > EPSILON) {
            if (printDeltas) {
                printf("Epsilon-err @%8d: %9.4f\t%9.4f\n", i, cpuVal, gpuVal);
            }
            diffCount++;
        }
    }

    mse /= numElements;

    if (optGpuTensorCpuPtr != nullptr) {
        *optGpuTensorCpuPtr = gpuTensorCpuMapped;
    } else {
        SAFE_FREE(gpuTensorCpuMapped);
    }
    return {diffCount, mse};
}
