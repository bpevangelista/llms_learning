#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>
#include "helpers.cu"


// One thread per output element [matSizeM, matSizeN] = [matSizeM, matSizeK] * [matSizeK, matSizeN]
// matA (row-major) and matB (col-major) allows continuous memory access
template <
    typename T, typename Acc,
    int32_t matSizeM, int32_t matSizeN, int32_t matSizeK        // Matrix A & B sizes
>
__global__ void gemm_kernel1x1(const T* __restrict__ matA, const T* __restrict__ matB, T* __restrict__ matOut) {

    // 1D thread group block in 1D dispatch grid
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    // Assumes output matrix (matSizeM x matSizeN) evenly divides by kernel size
    int32_t row = index / matSizeN;
    int32_t col = index % matSizeN;

    Acc acc = static_cast<Acc>(0.0f);
    for (int32_t k=0; k < matSizeK; k++) {
        // Row-major * Column-major
        acc += static_cast<Acc>(
            matA[row * matSizeK + k] *
            matB[col * matSizeK + k]);
    }

    matOut[index] = static_cast<T>(acc);
}


template <typename T, typename Acc>
int gemm_main(float EPSILON = 0.001f) {
    // MxK and NxK matrices
    constexpr int32_t matSizeM = 2048;      // 2k context
    constexpr int32_t matSizeK = 4096;      // 4k token dimension
    constexpr int32_t matSizeN = 2048;      // 2k context

    // Keep CPU copy of data for validation later
    T *cpuMatA, *cpuMatB, *cpuMatOut;

    // Alloc CUDA device memory with random data for matA, matB and zeros for matOut
    T* matA = deviceTensorRand<T>(1, matSizeM, matSizeK, 2.0f, &cpuMatA);     // [-2, 2] rand values
    T* matB = deviceTensorRand<T>(1, matSizeK, matSizeN, 2.0f, &cpuMatB);     // [-2, 2] rand values
    // Output is matRows x matRows
    T* matOut = deviceTensorRand<T>(1, matSizeM, matSizeN, 0.0f, &cpuMatOut); // Zeroed
    if (matA == nullptr || matB == nullptr || matOut == nullptr) {
        return -1; // error
    }

    // Empiric dispatch block size of 128 threads (rational, SM can dispatch 4xWarps of 32 threads)
    dim3 threadGroupSize = dim3(128, 1, 1);
    dim3 threadGroupsCount = dim3((matSizeM * matSizeN) / threadGroupSize.x);
    // For simplicity, matrix size must be multiple of block size
    assert((matSizeM * matSizeN) % threadGroupSize.x == 0);
    int32_t dynamicSharedMemSize = 0;

    // Calculate on GPU
    hipStream_t stream;
    hipEvent_t kernelStart, kernelStop;
    hipStreamCreate(&stream);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(kernelStart, stream);
    gemm_kernel1x1
        <T, Acc, matSizeM, matSizeN, matSizeK>
        <<<threadGroupsCount, threadGroupSize, dynamicSharedMemSize, stream>>>
        (matA, matB, matOut);
    hipEventRecord(kernelStop, stream);

    // Wait for GPU (just for correctness as CPU is much slower)
    hipError_t cudaStatus = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Calculate runtime (ideally avg over many runs)
    float kernelMs = 0.0f;
    hipEventElapsedTime(&kernelMs, kernelStart, kernelStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    printf("Kernel runtime: %.2fms\n", kernelMs);

#ifdef CPU_MATH_VALIDATION_ENABLED
    // Calculate on CPU
    for (int i=0; i<matSizeM; ++i) {
        for (int j=0; j<matSizeN; ++j) {
            Acc acc = 0.0f;
            for (int k=0; k<matSizeK; ++k) {
                float temp = static_cast<float>(cpuMatA[i * matSizeK + k]) * static_cast<float>(cpuMatB[j * matSizeK + k]);
                if (sizeof(Acc) == 2) { // half
                    acc = __float2half_rn(static_cast<float>(acc) + temp);
                } else { // float
                    acc += temp;
                }
            }
            cpuMatOut[i * matSizeN + j] = acc;
        }
    }

    // Validate CPU vs GPU computation
    auto [diffs, mse] = debugCompare<T>(cpuMatOut, matOut, nullptr, matSizeM * matSizeN, EPSILON);
    printf("Epsilon-diffs: count %d, perc %.3f, MSE %.4f\n", diffs, diffs/(float)(matSizeM * matSizeN), mse);
#endif

    SAFE_FREE(cpuMatA);
    SAFE_FREE(cpuMatB);
    SAFE_FREE(cpuMatOut);
    SAFE_CUDA_FREE(matA);
    SAFE_CUDA_FREE(matB);
    SAFE_CUDA_FREE(matOut);
    return 0;
}

int main() {
    //return gemm_main<float, float>(0.001f);    // 950ms on 3060TI for 2k-4k-gemm
    return gemm_main<half, float>(0.1f);       // 300ms on 3060TI for 2k-4k-gemm
    //return gemm_main<half, half>(0.75f);      // 360ms on 3060TI for 2k-4k-gemm
}