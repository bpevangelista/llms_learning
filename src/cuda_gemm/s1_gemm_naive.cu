#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include "helpers.cu"


// One thread per output element [matSizeM, matSizeN] = [matSizeM, matSizeK] * [matSizeK, matSizeN]
// matA (row-major) and matB (col-major) allows continuous memory access
__global__ void gemm_kernel1x1(const float* matA, const float* matB, float* matOut,
    const int32_t matSizeM, const int32_t matSizeN, const int32_t matSizeK) {

    // 1D thread group block in 1D dispatch grid
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    // Assumes output matrix (matSizeM x matSizeN) evenly divides by kernel size
    int32_t row = index / matSizeN;
    int32_t col = index % matSizeN;

    float acc = 0.0f;
    for (int32_t k=0; k < matSizeK; k++) {
        // Row-major * Column-major
        acc += matA[row * matSizeK + k] * matB[col * matSizeK + k];
    }

    matOut[index] = acc;
}


int main() {
    // MxK and NxK matrices
    constexpr int32_t matSizeM = 2048;      // 2k context
    constexpr int32_t matSizeK = 4096;      // 4k token dimension
    constexpr int32_t matSizeN = 2048;      // 2k context

    // Keep CPU copy of data for validation later
    float *cpuMatA, *cpuMatB, *cpuMatOut;

    // Alloc CUDA device memory with random data for matA, matB and zeros for matOut
    float* matA = deviceTensorRand<float>(1, matSizeM, matSizeK, 2.0f, &cpuMatA);       // [-2, 2] rand values
    float* matB = deviceTensorRand<float>(1, matSizeN, matSizeK, 2.0f, &cpuMatB);       // [-2, 2] rand values
    // Output is matRows x matRows
    float* matOut = deviceTensorRand<float>(1, matSizeM, matSizeN, 0.0f, &cpuMatOut);     // Zeroed
    if (matA == nullptr || matB == nullptr || matOut == nullptr) {
        return -1; // error
    }

    // Empiric dispatch block size of 128 threads (rational, SM can dispatch 4xWarps of 32 threads)
    dim3 threadGroupSize = dim3(128, 1, 1);
    dim3 threadGroupsCount = dim3((matSizeM * matSizeN) / threadGroupSize.x);
    // For simplicity, matrix size must be multiple of block size
    assert((matSizeM * matSizeN) % threadGroupSize.x == 0);
    int32_t dynamicSharedMemSize = 0;

    // Calculate GEMM on GPU
    hipStream_t stream;
    hipEvent_t kernelStart, kernelStop;
    hipStreamCreate(&stream);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(kernelStart, stream);
    gemm_kernel1x1<<<threadGroupsCount, threadGroupSize, dynamicSharedMemSize, stream>>>(
        matA, matB, matOut, matSizeM, matSizeN, matSizeK);
    hipEventRecord(kernelStop, stream);

    // Wait for GPU (just for correctness as CPU is much slower)
    hipError_t cudaStatus = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Calculate runtime (ideally avg over many runs)
    float kernelMs = 0.0f;
    hipEventElapsedTime(&kernelMs, kernelStart, kernelStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    printf("Kernel runtime: %.2fms\n", kernelMs); // 950ms on 3060TI for 2k-4k-gemm, MSE 0

#ifdef CPU_MATH_VALIDATION_ENABLED
    // Calculate GEMM on CPU
    for (int i=0; i<matSizeM; ++i) {
        for (int j=0; j<matSizeN; ++j) {
            float acc = 0.0f;
            for (int k=0; k<matSizeK; ++k) {
                acc = cpuMatA[i * matSizeK + k] * cpuMatB[j * matSizeK + k] + acc;
            }
            cpuMatOut[i * matSizeN + j] = acc;
        }
    }

    // Validate CPU vs GPU computation
    auto [diffs, mse] = debugCompare<float>(cpuMatOut, matOut, nullptr, matSizeM * matSizeN);
    printf("Epsilon-diffs: count %d, perc %.3f, MSE %.4f\n", diffs, diffs/(float)(matSizeM * matSizeN), mse);
#endif

    SAFE_FREE(cpuMatA);
    SAFE_FREE(cpuMatB);
    SAFE_FREE(cpuMatOut);
    SAFE_CUDA_FREE(matA);
    SAFE_CUDA_FREE(matB);
    SAFE_CUDA_FREE(matOut);
    return 0;
}