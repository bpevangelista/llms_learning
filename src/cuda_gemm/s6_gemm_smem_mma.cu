#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>
#include "helpers.cu"

__device__ __forceinline__ uint32_t lane_id() { uint32_t res; asm("mov.u32 %0, %laneid;" : "=r"(res) ); return res; }
__device__ float f1(uint32_t val) { return __half2float( ((half*)&val)[0] ); }
__device__ float f2(uint32_t val) { return __half2float( ((half*)&val)[1] ); }

__device__ void cp_async16B_cache_all(void *smem_ptr, const void *gmem_ptr, bool predicate=true) {
    constexpr int32_t kCopySize = 16;
    int32_t src_size = predicate? kCopySize : 0;
    uint32_t smem_ptr_u32 = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr)); // Cast smem_ptr to u32

    asm volatile("cp.async.ca.shared.global [%0], [%1], %2, %3;"
        :: "r"(smem_ptr_u32), "l"(gmem_ptr), "n"(kCopySize), "r"(src_size));
}

__device__ void cp_async16B_cache_global(void *smem_ptr, const void *gmem_ptr, bool predicate=true) {
    constexpr int32_t kCopySize = 16;
    int32_t src_size = predicate? kCopySize : 0;
    uint32_t smem_ptr_u32 = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr)); // Cast smem_ptr to u32

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2, %3;"
        :: "r"(smem_ptr_u32), "l"(gmem_ptr), "n"(kCopySize), "r"(src_size));
}

__device__ void cp_async_wait_all() {
    asm volatile("cp.async.wait_all;");
}


// One thread per output element [kMatSizeM, kMatSizeN] = [kMatSizeM, kMatSizeK] * [kMatSizeK, kMatSizeN]
// mat_a (row-major) and mat_b (col-major) allows continuous memory access
template <
    int32_t kMatSizeM, int32_t kMatSizeN, int32_t kMatSizeK                // Matrix A & B sizes
>
__global__ void gemm_kernel_m16n8k16(const half* __restrict__ mat_a, const half* __restrict__ mat_b, half* __restrict__ mat_out) {

    constexpr int32_t kKernelSizeMK = 16;
    constexpr int32_t kKernelSizeN = 8;

    // 2D thread group block in 2D dispatch grid
    int32_t row = blockIdx.x * kKernelSizeMK;
    int32_t col = blockIdx.y * kKernelSizeN;

    // Handle output matrix (kMatSizeM x kMatSizeN) not evenly divisible by kernel size
    if (row >= kMatSizeM || col >= kMatSizeN) {
        return; // Early exit
    }

    uint32_t laneid = lane_id();
    uint32_t shift_k = laneid >> 2;
    uint32_t shift_elem = laneid % 4;

    constexpr int32_t kBufferCount = 2; // Triple buffering
    __shared__ uint32_t smem_mat_a[kKernelSizeMK*kKernelSizeMK*kBufferCount/2];
    __shared__ uint32_t smem_mat_b[kKernelSizeMK*kKernelSizeN*kBufferCount/2];

    // MatOut 16x8 tile
    // Calculate 4xf32 p/thread
    float acc[4] = {};

    for (int32_t k=0; k < kMatSizeK; k += kKernelSizeMK) {
        // MatA 16x16 tile
        // 8xf16 across 4 32b register (2xf16 p/register)
        #pragma unroll
        for (int cp = 0; cp < 16; cp++) {
            uint32_t* mat_a_u32 = (uint32_t*)&mat_a[(row + cp) * kMatSizeK + k];
            cp_async16B_cache_global(&smem_mat_a[cp * 8 + 0], &mat_a_u32[0]); // copy 8x
            cp_async16B_cache_global(&smem_mat_a[cp * 8 + 4], &mat_a_u32[4]); // copy 8x
        }

        // MatB 16x8 tile
        // 4xf16 across 2 32b register (2xf16 p/register)
        #pragma unroll
        for (int cp = 0; cp < 8; cp++) {
            uint32_t* mat_b_u32 = (uint32_t*)&mat_b[(col + cp) * kMatSizeK + k];
            cp_async16B_cache_global(&smem_mat_b[cp * 8 + 0], &mat_b_u32[0]);
            cp_async16B_cache_global(&smem_mat_b[cp * 8 + 4], &mat_b_u32[4]);
        }

        cp_async_wait_all();

        uint32_t a0 = smem_mat_a[shift_k * 8 + shift_elem];
        uint32_t a1 = smem_mat_a[shift_k * 8 + shift_elem + 64];
        uint32_t a2 = smem_mat_a[shift_k * 8 + shift_elem + 4];
        uint32_t a3 = smem_mat_a[shift_k * 8 + shift_elem + 64 + 4];
        uint32_t b0 = smem_mat_b[shift_k * 8 + shift_elem];
        uint32_t b1 = smem_mat_b[shift_k * 8 + shift_elem + 4];

        asm volatile(
          "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
          "{%0,  %1,  %2,  %3},"
          "{%4,  %5,  %6,  %7}, {%8,  %9},"
          "{%10, %11, %12, %13};\n"
          : "=f"(acc[0]), "=f"(acc[1]), "=f"(acc[2]), "=f"(acc[3])
          :  "r"(a0),  "r"(a1),  "r"(a2),  "r"(a3), "r"(b0),  "r"(b1),
             "f"(acc[0]),  "f"(acc[1]),  "f"(acc[2]),  "f"(acc[3]));

#if 0 // Debug
        if (laneid == 0) {
            printf("\na %.3f %.3f %.3f %.3f %.3f %.3f %.3f %.3f", f1(a0), f2(a0), f1(a1), f2(a1), f1(a2), f2(a2), f1(a3), f2(a3));
            printf("\nb %.3f %.3f %.3f %.3f", f1(b0), f2(b0), f1(b1), f2(b1));
            printf("\nacc %.3f %.3f %.3f %.3f\n\n", acc[0], acc[1], acc[2], acc[3]);
        }
#endif
    }

    half* mat_out_top = &mat_out[(row + shift_k + 0) * kMatSizeN + col];
    half* mat_out_bot = &mat_out[(row + shift_k + 8) * kMatSizeN + col];
    mat_out_top[shift_elem * 2 + 0] = __float2half_rn(acc[0]);
    mat_out_top[shift_elem * 2 + 1] = __float2half_rn(acc[1]);
    mat_out_bot[shift_elem * 2 + 0] = __float2half_rn(acc[2]);
    mat_out_bot[shift_elem * 2 + 1] = __float2half_rn(acc[3]);
}


template <typename T, typename Acc>
int gemm_main(float EPSILON = 0.001f) {
    // MxK and NxK matrices
    constexpr int32_t kMatSizeM = 2048; // 2k context
    constexpr int32_t kMatSizeK = 4096; // 4k token dimension
    constexpr int32_t kMatSizeN = 2048; // 2k context

    // Per-Kernel Computation Size
    constexpr int32_t kKernelSizeMK = 16;
    constexpr int32_t kKernelSizeN = 8;

    // Keep CPU copy of data for validation later
    T *cpu_mat_a, *cpu_mat_b, *cpu_mat_out;

    // Alloc CUDA device memory with random data for mat_a, mat_b and zeros for mat_out
    T* mat_a = deviceTensorRand<T>(1, kMatSizeM, kMatSizeK, 2.0f, &cpu_mat_a);     // [-2, 2] rand values
    T* mat_b = deviceTensorRand<T>(1, kMatSizeK, kMatSizeN, 2.0f, &cpu_mat_b);     // [-2, 2] rand values
    // Output is matRows x matRows
    T* mat_out = deviceTensorRand<T>(1, kMatSizeM, kMatSizeN, 0.0f, &cpu_mat_out); // [0, 0] Zero it
    if (mat_a == nullptr || mat_b == nullptr || mat_out == nullptr) {
        return -1; // error
    }

    // Must be warp-size (32) multiple due to mma instruction
    dim3 thread_group_size = dim3(32, 1, 1);
    dim3 thread_groups = dim3(
        CEIL_DIV(kMatSizeM, kKernelSizeMK),
        CEIL_DIV(kMatSizeN, kKernelSizeN));
    int32_t dynamic_smem_size = 0;

    // Calculate on GPU
    hipStream_t stream;
    hipEvent_t kernel_start, kernel_stop;
    hipStreamCreate(&stream);
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);

    hipEventRecord(kernel_start, stream);
    gemm_kernel_m16n8k16
        <kMatSizeM, kMatSizeN, kMatSizeK>
        <<<thread_groups, thread_group_size, dynamic_smem_size, stream>>>
        (mat_a, mat_b, mat_out);
    hipEventRecord(kernel_stop, stream);

    // Wait for GPU (just for correctness as CPU is much slower)
    hipError_t status = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Calculate runtime (ideally avg over many runs)
    float kernel_elapsed_ms = 0.0f;
    hipEventElapsedTime(&kernel_elapsed_ms, kernel_start, kernel_stop);
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
    printf("Kernel runtime: %.2fms\n", kernel_elapsed_ms);

#ifdef CPU_MATH_VALIDATION_ENABLED
    // Calculate on CPU
    for (int i=0; i < kMatSizeM; ++i) {
        for (int j=0; j < kMatSizeN; ++j) {
            Acc acc = 0.0f;
            for (int k=0; k < kMatSizeK; ++k) {
                float temp = static_cast<float>(cpu_mat_a[i * kMatSizeK + k]) * static_cast<float>(cpu_mat_b[j * kMatSizeK + k]);
                if (sizeof(Acc) == 2) { // half
                    acc = __float2half_rn(static_cast<float>(acc) + temp);
                } else { // float
                    acc += temp;
                }
            }
            cpu_mat_out[i * kMatSizeN + j] = acc;
        }
    }

    // Validate CPU vs GPU computation
    T* mat_out_cpu_copied;
    auto [diffs, mse] = debugCompare(cpu_mat_out, mat_out, &mat_out_cpu_copied, kMatSizeM * kMatSizeN, EPSILON);
    printf("Epsilon-diffs: count %d, perc %.3f, MSE %.4f\n", diffs, diffs/(float)(kMatSizeM * kMatSizeN), mse);

    // Debug small matrices
    if (kMatSizeM <= 32 && kMatSizeN <= 32) {
        printTensor("cpu_mat_a\n", cpu_mat_a, kMatSizeM, kMatSizeK);
        printTensor("cpu_mat_b\n", cpu_mat_b, kMatSizeN, kMatSizeK);
        printTensor("cpu_mat_out\n", cpu_mat_out, kMatSizeM, kMatSizeN);
        printTensor("cuda_mat_out\n", mat_out_cpu_copied, kMatSizeM, kMatSizeN);
    }
    SAFE_FREE(mat_out_cpu_copied);
#endif

    SAFE_FREE(cpu_mat_a);
    SAFE_FREE(cpu_mat_b);
    SAFE_FREE(cpu_mat_out);
    SAFE_CUDA_FREE(mat_a);
    SAFE_CUDA_FREE(mat_b);
    SAFE_CUDA_FREE(mat_out);
    return 0;
}

int main() {
    return gemm_main<half, float>(0.1f);       // 6.2ms on 3060TI for 2k-4k-gemm, MSE 0.0007 or 0.02 (k=32)
}