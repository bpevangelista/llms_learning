#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>
#include "helpers.cu"

#include <cutlass/cutlass.h>

// Parallel on output index [Rows, Rows] = [Rows, Cols] * [Cols, Rows]
// matA is row-major and matB col-major for continuous memory access
template <typename T, typename Acc>
__global__ void gemm_kernel(const T* __restrict__ matA, const T* __restrict__ matB, T* __restrict__ matOut, const int32_t matRows, const int32_t matCols) {
    // Using 1D block
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    // Out is Rows x Rows
    int32_t row = index / matRows;
    int32_t col = index % matRows;

    Acc acc = static_cast<Acc>(0.0f);
    for (int32_t k=0; k < matCols; k++) {
        // Row-major * Column-major
        acc = static_cast<Acc>(matA[row * matCols + k] * matB[col * matCols + k]) + acc;
    }

    matOut[index] = static_cast<T>(acc);
}


template <typename T, typename Acc>
int gemm_main(float EPSILON = 0.001f) {
     constexpr int32_t matRows = 2048; // 2k context
     constexpr int32_t matCols = 4096; // 4k token dimension

    // Keep CPU copy of data for validation later
    T *cpuMatA, *cpuMatB, *cpuMatOut;

    // Alloc CUDA device memory with random data for matA, matB and zeros for matOut
    T* matA = deviceTensorRand<T>(1, matRows, matCols, 2.0f, &cpuMatA);     // [-2, 2] rand values
    T* matB = deviceTensorRand<T>(1, matCols, matRows, 2.0f, &cpuMatB);     // [-2, 2] rand values
    // Output is matRows x matRows
    T* matOut = deviceTensorRand<T>(1, matRows, matRows, 0.0f, &cpuMatOut); // [ 0, 0] rand values
    if (matA == nullptr || matB == nullptr || matOut == nullptr) {
        return -1; // error
    }

    // Empiric block size of 128 threads (rational, SM can dispatch 4xWarps of 32 threads)
    dim3 blockSize = dim3(128, 1, 1);
    dim3 blocksCount = dim3(ceil(matRows * matRows / float(blockSize.x)));
    int32_t sharedMemorySize = 0;

    // Calculate on GPU
    hipStream_t stream;
    hipEvent_t kernelStart, kernelStop;
    hipStreamCreate(&stream);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(kernelStart, 0);
    gemm_kernel<T, Acc><<<blocksCount, blockSize, sharedMemorySize, stream>>>(matA, matB, matOut, matRows, matCols);
    hipEventRecord(kernelStop, 0);

    // Calculate on CPU
    for (int i=0; i<matRows; ++i) {
        for (int j=0; j<matRows; ++j) {
            Acc acc = 0.0f;
            for (int k=0; k<matCols; ++k) {
                float temp = static_cast<float>(cpuMatA[i * matCols + k]) * static_cast<float>(cpuMatB[j * matCols + k]);
                if (sizeof(Acc) == 2) { // half
                    acc = __float2half_rn(static_cast<float>(acc) + temp);
                } else { // float
                    acc += temp;
                }
            }
            cpuMatOut[i * matRows + j] = acc;
        }
    }

    // Wait for GPU (just for correctness as CPU is much slower)
    hipError_t cudaStatus = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Calculate runtime (ideally avg over many runs)
    float kernelMs = 0.0f;
    hipEventElapsedTime(&kernelMs, kernelStart, kernelStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    printf("Kernel runtime: %.2fms\n", kernelMs);

    // Validate CPU vs GPU computation
    debugCompareAndPrint(cpuMatOut, matOut, matRows * matRows, EPSILON);

    // Debug
    //printMat(cpuMatA, matRows, matCols);
    //printMat(cpuMatB, matRows, matCols);
    //printMat(cpuMatOut, matRows, matRows);
    //printMat(gpuToCpuMatOut, matRows, matRows);

    SAFE_FREE(cpuMatA);
    SAFE_FREE(cpuMatB);
    SAFE_FREE(cpuMatOut);
    SAFE_CUDA_FREE(matA);
    SAFE_CUDA_FREE(matB);
    SAFE_CUDA_FREE(matOut);
    return 0;
}

int main() {
    //return gemm_main<float, float>(0.001);    // 950ms on 3060TI for 2k-4k-gemm
    return gemm_main<half, float>(0.25f);       // 300ms on 3060TI for 2k-4k-gemm
    //return gemm_main<half, half>(0.75f);      // 360ms on 3060TI for 2k-4k-gemm
}