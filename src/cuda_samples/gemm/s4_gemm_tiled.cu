#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>
#include "helpers.cu"

// One thread per output element [matSizeM, matSizeN] = [matSizeM, matSizeK] * [matSizeK, matSizeN]
// matA (row-major) and matB (col-major) allows continuous memory access
template <
    typename T, typename Acc,
    int32_t matSizeM, int32_t matSizeN, int32_t matSizeK,               // Matrix A & B sizes
    int32_t kernelSizeM, int32_t kernelSizeN                            // Kernel computation size
>
__global__ void gemm_kernel_mn(const T* __restrict__ matA, const T* __restrict__ matB, T* __restrict__ matOut) {

    // 2D thread group block in 2D dispatch grid
    int32_t row = (blockIdx.x * blockDim.x + threadIdx.x) * kernelSizeM;
    int32_t col = (blockIdx.y * blockDim.y + threadIdx.y) * kernelSizeN;
    // Handle output matrix (matSizeM x matSizeN) not evenly divisible by kernel size
    if (row >= matSizeM || col >= matSizeN) {
        return; // Early exit
    }

    // Accumulator registers
    constexpr int32_t kernelSizeK = 16;
    Acc acc[kernelSizeM*kernelSizeN] = {0};

    #pragma unroll // Too many registers?
    for (int32_t k=0; k < matSizeK; k += kernelSizeK) {
        // Compute kernelSizeM x kernelSizeN x kernelSizeK
        for (int ki=0; ki < kernelSizeM; ki++) {
            for (int kj=0; kj < kernelSizeN; kj++) {
                for (int kk=0; kk < kernelSizeK; kk++) {
                    acc[ki * kernelSizeN + kj] += static_cast<Acc>(
                        matA[(row + ki) * matSizeK + k + kk] *
                        matB[(col + kj) * matSizeK + k + kk]);
                }
            }
        }
    }

    #pragma unroll
    for (int i=0; i < kernelSizeM; i++) {
        for (int j=0; j < kernelSizeN; j++) {
            matOut[(row + i) * matSizeN + col + j] = acc[i * kernelSizeN + j];
        }
    }
}


template <typename T, typename Acc>
int gemm_main(float EPSILON = 0.001f) {
    // MxK and NxK matrices
    constexpr int32_t matSizeM = 2048; // 2k context
    constexpr int32_t matSizeK = 4096; // 4k token dimension
    constexpr int32_t matSizeN = 2048; // 2k context
    // Per-Kernel Computation Size
    constexpr int32_t kernelSizeM = 16;
    constexpr int32_t kernelSizeN = 8;

    // Keep CPU copy of data for validation later
    T *cpuMatA, *cpuMatB, *cpuMatOut;

    // Alloc CUDA device memory with random data for matA, matB and zeros for matOut
    T* matA = deviceTensorRand<T>(1, matSizeM, matSizeK, 2.0f, &cpuMatA);     // [-2, 2] rand values
    T* matB = deviceTensorRand<T>(1, matSizeK, matSizeN, 2.0f, &cpuMatB);     // [-2, 2] rand values
    // Output is matRows x matRows
    T* matOut = deviceTensorRand<T>(1, matSizeM, matSizeN, 0.0f, &cpuMatOut); // [ 0, 0] rand values
    if (matA == nullptr || matB == nullptr || matOut == nullptr) {
        return -1; // error
    }

    // Empiric dispatch block size 8x16 == 128 threads (match previous sample)
    dim3 threadGroupSize = dim3(8, 16, 1);
    dim3 threadGroupsCount = dim3(
        CEIL_DIV(matSizeM, threadGroupSize.x * kernelSizeM),
        CEIL_DIV(matSizeN, threadGroupSize.y * kernelSizeN));
    int32_t dynamicSharedMemSize = 0; // Still not using it

    // Calculate on GPU
    hipStream_t stream;
    hipEvent_t kernelStart, kernelStop;
    hipStreamCreate(&stream);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(kernelStart, 0);
    gemm_kernel_mn
        <T, Acc, matSizeM, matSizeN, matSizeK, kernelSizeM, kernelSizeN>
        <<<threadGroupsCount, threadGroupSize, dynamicSharedMemSize, stream>>>
        (matA, matB, matOut);
    hipEventRecord(kernelStop, 0);

    // Calculate on CPU
    for (int i=0; i<matSizeM; ++i) {
        for (int j=0; j<matSizeN; ++j) {
            Acc acc = 0.0f;
            for (int k=0; k<matSizeK; ++k) {
                float temp = static_cast<float>(cpuMatA[i * matSizeK + k]) * static_cast<float>(cpuMatB[j * matSizeK + k]);
                if (sizeof(Acc) == 2) { // half
                    acc = __float2half_rn(static_cast<float>(acc) + temp);
                } else { // float
                    acc += temp;
                }
            }
            cpuMatOut[i * matSizeN + j] = acc;
        }
    }

    // Wait for GPU (just for correctness as CPU is much slower)
    hipError_t cudaStatus = hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    // Calculate runtime (ideally avg over many runs)
    float kernelMs = 0.0f;
    hipEventElapsedTime(&kernelMs, kernelStart, kernelStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    printf("Kernel runtime: %.2fms\n", kernelMs);

    // Validate CPU vs GPU computation
    T* matOutCpuPtr;
    auto [diffs, mse] = debugCompare(cpuMatOut, matOut, &matOutCpuPtr, matSizeM * matSizeN, EPSILON);
    printf("Epsilon-diffs: count %d, perc %.3f. MSE %.4f\n", diffs, diffs/(float)(matSizeM * matSizeN), mse);

    // Debug small matrices
    if (matSizeM <= 32 && matSizeN <= 32) {
        printTensor(cpuMatA, matSizeM, matSizeK);
        printTensor(cpuMatB, matSizeN, matSizeK);
        printTensor(cpuMatOut, matSizeM, matSizeN);
        printTensor(matOutCpuPtr, matSizeM, matSizeN);
    }

    SAFE_FREE(cpuMatA);
    SAFE_FREE(cpuMatB);
    SAFE_FREE(cpuMatOut);
    SAFE_FREE(matOutCpuPtr);
    SAFE_CUDA_FREE(matA);
    SAFE_CUDA_FREE(matB);
    SAFE_CUDA_FREE(matOut);
    return 0;
}

int main() {
    //return gemm_main<float, float>(0.001);    // 100ms on 3060TI for 2k-4k-gemm, MSE 0
    return gemm_main<half, float>(0.1f);       // 100ms on 3060TI for 2k-4k-gemm, MSE 0.0007 or 0.02 (k=32)
}